#include "hip/hip_runtime.h"
#include <stdint.h>
#include <sys/time.h>
#include <fstream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>

using namespace std;

#define NUM_SMS 110
// must be power of two
#define BLOCK_SIZE 256
// must be power of two
#define NUM_THREADS_PER_SM 2048
#define NUM_BLOCKS_PER_SM (NUM_THREADS_PER_SM / BLOCK_SIZE)
#define NUM_BLOCKS (NUM_SMS * NUM_BLOCKS_PER_SM)
#define NUM_THREADS (NUM_THREADS_PER_SM * NUM_SMS)

#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define MAX(x, y) (((x) > (y)) ? (x) : (y))

#define BATCH_SIZE 10
#define TREE_DEPTH 4
#define TREE_SIZE ((1 << TREE_DEPTH) - 1)
#define NUM_TREES 60

typedef uint8_t uint1_t;

typedef struct {
  uint16_t field;
  uint16_t split;
} node;

__global__ void run(uint32_t *input_full, uint32_t input_count, uint32_t *output_full, uint32_t *output_count) {
  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t *input_buf = input_full + index * input_count;
  uint32_t *output_buf = output_full + index * input_count;

  uint32_t input_idx = 0;
  uint32_t output_buf_idx = 0;
  uint16_t words_consumed = 0;
  uint16_t buffer[BATCH_SIZE];
  __shared__ node tree[TREE_SIZE * NUM_TREES];

  if (threadIdx.x == 0) {
    for (int i = 0; i < TREE_SIZE * NUM_TREES; i++) {
      tree[i].field = i % BATCH_SIZE;
      tree[i].split = 1 << 15; // midpoint of 16-bit values
    }
  }
  __syncthreads();


  #define BUF_SIZE 256
  uint32_t input[BUF_SIZE];
  uint32_t output[(BUF_SIZE + BATCH_SIZE - 1) / BATCH_SIZE]; // max outputs for BUF_SIZE inputs
  for (uint32_t ii = input_idx; ii < input_count; ii += BUF_SIZE) {
    for (uint32_t i = ii; i < MIN(input_count, ii + BUF_SIZE); i++) {
      input[i - ii] = input_buf[i];
    }
    uint32_t output_idx = 0;
    for (uint32_t i = 0; i < MIN(input_count - ii, BUF_SIZE); i++) {
      buffer[words_consumed++] = input[i];
      if (words_consumed == BATCH_SIZE) {
        uint32_t sum = 0;
        for (uint32_t tree_base = 0; tree_base < TREE_SIZE * NUM_TREES; tree_base += TREE_SIZE) {
          int tree_idx = 0;
          while (tree_idx < TREE_SIZE / 2) {
            node cur = tree[tree_base + tree_idx];
            tree_idx = buffer[cur.field] < cur.split ? tree_idx * 2 + 1 : tree_idx * 2 + 2;
          }
          sum += tree[tree_base + tree_idx].split;
        }
        output[output_idx++] = sum;
        words_consumed = 0;
      }
    }
    for (uint32_t i = 0; i < output_idx; i++) {
      output_buf[output_buf_idx + i] = output[i];
    }
    output_buf_idx += output_idx;
  }
  *(output_count + index) = output_buf_idx;
}

int main(int argc, char **argv) {
  uint32_t GLOBAL_CHARS = 10000000;
  uint32_t CHARS = atoi(argv[1]);

  // extracts "ad_id" and "ad_type"
  uint8_t seq_confs[] = {1, 34, 2, 97, 3, 100, 4, 95, 5, 105, 6, 100, 200, 34, 8, 121, 9, 112, 10, 101, 200, 34};
  uint8_t split_confs[] = {4, 7, 116};

  ifstream infile("kafka-json.txt");
  string line;

  uint32_t conf_size = sizeof(seq_confs) + sizeof(split_confs);
  uint32_t input_buf_size = conf_size + GLOBAL_CHARS;
  uint8_t *input_buf = new uint8_t[input_buf_size];
  uint32_t global_chars = 0;
  uint64_t chars = 0;
  memcpy(input_buf + global_chars, seq_confs, sizeof(seq_confs));
  global_chars += sizeof(seq_confs);
  memcpy(input_buf + global_chars, split_confs, sizeof(split_confs));
  global_chars += sizeof(split_confs);
  while (getline(infile, line)) {
    if (chars == 0 && global_chars + line.length() > conf_size + CHARS) {
      chars = global_chars;
    }
    if (global_chars + line.length() > input_buf_size) {
      break;
    }
    memcpy(input_buf + global_chars, line.c_str(), line.length());
    global_chars += line.length();
  }

  chars = chars / 4 * 4;
  uint8_t *combined_input = new uint8_t[chars * NUM_THREADS];
  for (uint64_t i = 0; i < NUM_THREADS; i++) {
    memcpy(combined_input + i * chars, input_buf + i * 10, chars);
    uint32_t *slice = (uint32_t *)(combined_input + i * chars);
    for (uint32_t j = 0; j < chars / 4; j++) {
      slice[j] = slice[j] & ((1 << 16) - 1);
    }
  }

  uint8_t *output_buf = new uint8_t[chars];
  uint32_t output_count;

  uint8_t *output_dev, *input_dev;
  uint32_t *output_count_dev;
  hipSetDevice(0);
  assert(hipMalloc((void **) &output_dev, chars * NUM_THREADS) == hipSuccess);
  assert(hipMalloc((void **) &input_dev, chars * NUM_THREADS) == hipSuccess);
  assert(hipMalloc((void **) &output_count_dev, sizeof(uint32_t) * NUM_THREADS) == hipSuccess);

  hipMemcpy(input_dev, combined_input, chars * NUM_THREADS, hipMemcpyHostToDevice);

  struct timeval start, end, diff;
  gettimeofday(&start, 0);
  run<<<NUM_BLOCKS, BLOCK_SIZE>>>((uint32_t *)input_dev, chars / 4, (uint32_t *)output_dev, output_count_dev);
  hipDeviceSynchronize();
  gettimeofday(&end, 0);
  timersub(&end, &start, &diff);
  hipMemcpy(&output_count, output_count_dev, sizeof(uint32_t), hipMemcpyDeviceToHost);
  hipMemcpy(output_buf, output_dev, output_count / 4, hipMemcpyDeviceToHost);
  double secs = diff.tv_sec + diff.tv_usec / 1000000.0;
  printf("%.2f Mops/s\n", ((chars * NUM_THREADS) + (chars * NUM_THREADS) / BATCH_SIZE * TREE_DEPTH * NUM_TREES) / 1000000.0 / secs);
  printf("%.2f MB/s, %d input tokens, %d output tokens, random output byte: %d\n",
    (chars * NUM_THREADS) / 1000000.0 / secs, (int) chars / 4, output_count,
    output_count == 0 ? 0 : output_buf[rand() % (output_count * 4)]);
  return 0;
}
